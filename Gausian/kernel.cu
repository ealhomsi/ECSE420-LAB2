#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "A.h"
#include "B.h"
#include "rational.h"

#include <string.h>
#include <stdio.h>


__device__ void printMatrix(double* matrix, double* b, unsigned dimension)
{
	printf("{\r\n");
	for (unsigned row = 0; row < dimension; row++)
	{
		printf("   {");
		for (unsigned column = 0; column < dimension; column++)
		{
			printf(" %.2f", matrix[column + row * dimension]);
		}
		printf("| %.2f", b[row]);
		printf(" }\r\n");
	}

	printf("}\r\n");
}

__device__ bool isCloseToZero(rational_t value)
{
	return value.numerator == 0;
}


__global__ void gaussianEliminationKernel(rational_t* matrix, unsigned dimension, rational_t* b, rational_t* x, bool* isSingular)
{
	__shared__ int swapWith;

	unsigned responsibleRow = blockDim.x * blockIdx.x + threadIdx.x;

	for (unsigned pivotRow = 0; pivotRow < dimension; pivotRow++)
	{
		if (responsibleRow == pivotRow)
		{
			rational_t pivot;
			swapWith = pivotRow - 1;

			do
			{
				swapWith++;
				pivot = matrix[swapWith * dimension + pivotRow];
			} while (isCloseToZero(pivot) && swapWith < dimension);

			if (swapWith < dimension)
			{
				for (unsigned col = pivotRow; col < dimension; col++)
				{
					matrix[col + swapWith * dimension] = rational_divide(matrix[col + swapWith * dimension], pivot);
				}

				b[swapWith] = rational_divide(b[swapWith], pivot);
			}
			else
			{
				*isSingular = true;
			}

			//printMatrix(matrix, b, dimension);
		}

		__syncthreads();

		if (swapWith >= dimension)
		{
			return;
		}
		else if (swapWith != pivotRow)
		{
			// Swapping phase, each thread is responsible for one column
			rational_t temp = matrix[threadIdx.x + pivotRow * dimension];
			matrix[threadIdx.x + pivotRow * dimension] = matrix[threadIdx.x + swapWith * dimension];
			matrix[threadIdx.x + swapWith * dimension] = temp;

			// Thread 0 is responsible for b

			if (threadIdx.x == 0)
			{
				rational_t temp = b[pivotRow];
				b[pivotRow] = b[swapWith];
				b[swapWith] = temp;
			}

			__syncthreads();

			/*if (threadIdx.x == 0)
			{
				printMatrix(matrix, b, dimension);
			}*/

			__syncthreads();
		}

		if (responsibleRow != pivotRow)
		{
			rational_t leadingValue = matrix[pivotRow + responsibleRow * dimension];

			for (unsigned col = pivotRow; col < dimension; col++)
			{
				matrix[col + responsibleRow * dimension] = rational_subtract(matrix[col + responsibleRow * dimension], rational_multiply(leadingValue, matrix[col + pivotRow * dimension]));
			}

			b[responsibleRow] = rational_subtract(b[responsibleRow], rational_multiply(leadingValue, b[pivotRow]));
		}

		__syncthreads();
	}

	/*if (threadIdx.x == 0)
	{
		printMatrix(matrix, b, dimension);
	}*/

	x[responsibleRow] = b[responsibleRow];
}

int main()
{
	const unsigned matrixSize = (sizeof(A) / sizeof(double));

	rational_t* a = new rational_t[matrixSize * matrixSize];
	// convert double A to rational_t a
	for (int i = 0; i < matrixSize; i++) {
		for (int j = 0; j < matrixSize; j++) {
			a[j + i * matrixSize] = rational_init(A[i][j]);
		}
	}

	rational_t* b = new rational_t[matrixSize];
	// convert double B to rational_t b
	for (int i = 0; i < matrixSize; i++) {
		b[i] = rational_init(B[i][0]);
	}

	rational_t* x = new rational_t[matrixSize];
	// init rational_t x
	for (int i = 0; i < matrixSize; i++) {
		x[i] = rational_init(0.0);
	}

	rational_t* deviceMatrix;
	rational_t* deviceB;
	rational_t* deviceX;
	bool* deviceSingular;

	hipMalloc((void**)& deviceMatrix, sizeof(rational_t) * matrixSize * matrixSize);
	hipMalloc((void**)& deviceB, sizeof(rational_t) * matrixSize);
	hipMalloc((void**)& deviceX, sizeof(rational_t) * matrixSize);
	hipMalloc((void**)& deviceSingular, sizeof(bool));

	hipMemcpy(deviceMatrix, a, sizeof(rational_t) * matrixSize * matrixSize, hipMemcpyHostToDevice);
	hipMemcpy(deviceB, b, sizeof(rational_t) * matrixSize, hipMemcpyHostToDevice);

	gaussianEliminationKernel << <1, matrixSize >> > (deviceMatrix, matrixSize, deviceB, deviceX, deviceSingular);

	bool singular;

	hipError_t cudaStatus = hipDeviceSynchronize();

	// Add vectors in parallel.
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	hipMemcpy(&singular, deviceSingular, sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(x, deviceX, matrixSize * sizeof(double), hipMemcpyDeviceToHost);

	if (singular)
	{
		// this hsould never happen
		printf("The matrix is not invertible, there is no unique solution.");
	}
	else
	{
		printf("\n{");
		for (int i = 0; i < matrixSize; i++) {
			rational_t item = x[i];
			printf("%.2f ", get_value(item));
		}
		printf("}\n");

	}

	hipFree(deviceSingular);
	hipFree(deviceX);
	hipFree(deviceMatrix);
	hipFree(deviceB);
	delete[] x;

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}